#include<hip/hip_runtime.h>
#include<stdio.h>

#define ceil_x_over_y(x,y) (x/y + ((x%y)?1:0))

__global__ void matrix(double *input, double num, int h, int w){
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < h*w){
        input[i] = num;
    }
}

__global__ void conv2d(double *input, double *kernel, double *output, int in_h, int in_w, int ker_h, int ker_w, int stride){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    i = i * stride;
    j = j * stride;
    
    int out_w = (in_w - ker_w) / stride + 1;
    if(i < in_h && j < in_w){
        double sum = 0;
        for(int k = 0; k < ker_h; k++){
            for(int l = 0; l < ker_w; l++){
                sum += input[(i + k) * in_w + j + l] * kernel[k * ker_w + l];
            }
        }
        output[i/stride + j/stride * out_w] = sum;
    }
}

__global__ void padding(double *input, double *output, int in_h, int in_w, int left, int right){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i < in_h && j < in_w){
        output[(i + left) * (in_w + left + right) + (j + left)] = input[i * in_w + j];
    }
}

class Conv{
public:
    Conv(int in_h, int in_w, int ker_h, int ker_w, int stride = 1, char pad = 'V'){
        hipError_t err;
        this->stride = stride;
        this->pad = pad;
        this->ker_h = ker_h;
        this->ker_w = ker_w;
        this->in_h = in_h;
        this->in_w = in_w;
        double *weights, *bias;
        err = hipMalloc((double **)&weights, sizeof(double) * ker_h * ker_w);
        if(err != hipSuccess)printf("Weights malloc error\n");
        err = hipMalloc((double **)&bias, sizeof(double));
        if(err != hipSuccess)printf("Bias malloc error\n");
        matrix<<<1024, 1>>>(weights, 1, ker_h * ker_w, 1);
        matrix<<<1024, 1>>>(bias, 0.1, 1, 1);
        this->weights = weights;
        this->bias = bias;
        if(this->pad == 'S'){
            int left = (ker_w-1) / 2;
            int right = ker_w - 1 - left;
            hipError_t err = hipMalloc((double **)&pad_input, sizeof(double) * (in_h + ker_h - 1) * (in_w + ker_w - 1));
            if(err != hipSuccess)printf("Pad input malloc error\n");
            this->left_up = left;
            this->right_down = right;
            this->out_w = ceil_x_over_y(in_w, stride);
            this->out_h = ceil_x_over_y(in_h, stride);
        }
        else if(this->pad == 'V'){
            hipError_t err = hipMalloc((double **)&pad_input, sizeof(double) * in_h * in_w);
            this->left_up = 0;
            this->right_down = 0;
            this->out_w = (in_w - ker_w) / stride + 1;
            this->out_h = (in_h - ker_h) / stride + 1;
        }
        double *output;
        err = hipMalloc((double **)&output, sizeof(double) * out_h * out_w);
        if(err != hipSuccess)printf("Output malloc error\n");
        this->output = output;
    }

    ~Conv(){
        hipFree(weights);
        hipFree(bias);
        hipFree(pad_input);
        hipFree(output);
    }

    double* forward(double *input){
        int num_elements = (this->in_h + this->ker_h - 1) * (this->in_w + this->ker_w - 1);
        dim3 dimGrid(16, 16);
        dim3 dimBlock(num_elements / dimGrid.x, num_elements / dimGrid.y);
        matrix<<<1024, 1>>>(this->pad_input, 0, this->in_h + this->ker_h - 1, this->in_w + this->ker_w - 1);
        padding<<<dimBlock, dimGrid>>>(input, this->pad_input, this->in_h, this->in_w, this->left_up, this->right_down);
        
        
        conv2d<<<dimBlock, dimGrid>>>(this->pad_input, this->weights, this->output, in_h + ker_h - 1, in_w + ker_w - 1,
                                        this->ker_h, this->ker_w, this->stride);
        return this->output;
    }

protected:
    double *weights;
    double *bias;
    double *pad_input;
    double *output;
    int stride;
    char pad;
    int ker_h;
    int ker_w;
    int in_h;
    int in_w;
    int left_up;
    int right_down;
    int out_w;
    int out_h;
};


int main()
{
    hipError_t err;
    double *mat, *kernel;// *output, *padded;
    int in_h = 4, in_w = 4;
    int ker_h = 2, ker_w = 2;
    Conv layer(in_h, in_w, ker_h, ker_w, 2, 'S');
    err = hipMalloc((double **)&mat, sizeof(double) * in_h * in_w);
    if(err != hipSuccess)printf("Input malloc error\n");
    err = hipMalloc((double **)&kernel, sizeof(double) * ker_h * ker_w);
    if(err != hipSuccess)printf("Kernel malloc error\n");

    matrix<<<1024, 1>>>(mat, 2, in_h * in_w, 1);
    matrix<<<1024, 1>>>(kernel, 1, ker_h * ker_w, 1);
    layer.forward(mat);

    return 0;
}