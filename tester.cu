#include "hip/hip_runtime.h"
#include<stdio.h>
#include "headers/Constant.cuh"
#include "headers/Dense.cuh"
#include "headers/Conv.cuh"
#include "headers/L1.cuh"
#include "headers/L2.cuh"
#include "headers/SGD.cuh"

#define reg_type L1 // l1, l2
#define DEBUG_DENSE

__global__ void fillMatrix(double *input, double num, int h, int w){
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < h*w){
        input[i] = i;
    }
}
#ifndef DEBUG_DENSE
void RegTestNorm(double alpha = 0.5, int num_weights = 5, double init_weights = 1);
void RegTestGrad(double alpha = 0.5, int num_weights = 5, double init_weights = 1);
void SGDTest(double momentum = 0, int num_weights = 5, double init_weights = 1, double init_grad = 0.3, double lr = 0.1, double reg_alpha = 0.5);
void InitializerTest(double init_weights = 1);
#endif
void DenseTest();

int main()
{
    DenseTest();
    return 0;
}
#ifndef DEBUG_DENSE
void InitializerTest(double init_weights){
    Constant initializer(init_weights);
    int size = 20;
    double *input = new double[size];
    memset(input, 0, size*sizeof(double));
    double* dev_w = initializer.initialize(size);
    hipMemcpy(input, dev_w, size*sizeof(double), hipMemcpyDeviceToHost);
    for(int i = 0; i < size; i++){
        printf("%f ", input[i]);
    }
    printf("\n");
}

void SGDTest(double momentum, int num_weights, double init_weights, double init_grad, double lr, double reg_alpha){
    L1 *reg = new L1(reg_alpha);
    SGD optimizer(lr, num_weights, 0.5, reg);
    fillMatrix<<<32,1>>>(optimizer.v, 1, num_weights, 1);
    double *weights, *gradients;
    double *host_weights = (double*)malloc(num_weights*sizeof(double));
    hipError_t err;
    err = hipMalloc((double **)&weights, num_weights * sizeof(double));
    printf(err == hipSuccess ? "." : "Memory allocation failed\n");
    err = hipMalloc((double **)&gradients, num_weights * sizeof(double));
    printf(err == hipSuccess ? "." : "Memory allocation failed\n");
    fillMatrix<<<16,1>>>(weights, init_weights, num_weights, 1);
    fillMatrix<<<16,1>>>(gradients, init_grad, num_weights, 1);
    optimizer.step(weights, gradients);
    err = hipMemcpy(host_weights, weights, num_weights * sizeof(double), hipMemcpyDeviceToHost);
    printf(err == hipSuccess ? "." : "Memory allocation failed1asd\n");
    printf("\n");
    for(int i = 0; i < num_weights; i++){
        printf("%f ", host_weights[i]);
    }
    printf("\n");
    free(host_weights);
    hipFree(weights);
    hipFree(gradients);
}

void RegTestNorm(double alpha, int num_weights, double init_weights){
    reg_type regularizer(alpha);
    double *pseudoWeights;
    double *norm, *host_norm;
    host_norm = (double*)malloc(sizeof(double)*1);
    hipError_t err;
    err = hipMalloc((double **)&pseudoWeights, sizeof(double)*num_weights);
    printf(err == hipSuccess ? "." : "Memory allocation failed\n");
    fillMatrix<<<num_weights+1, 1>>>(pseudoWeights, init_weights, num_weights, 1);
    err = hipMalloc((double **)&norm, sizeof(double));
    printf(err == hipSuccess ? "." : "Memory allocation failed\n");
    norm = regularizer.norm(pseudoWeights, num_weights);
    err = hipMemcpy(host_norm, norm, sizeof(double), hipMemcpyDeviceToHost);
    printf(err == hipSuccess ? "." : "Memory copy failed\n");
    printf("\n\n%f\n", *host_norm);
    free(host_norm);
    hipFree(pseudoWeights);
    hipFree(norm);
}

void RegTestGrad(double alpha, int num_weights, double init_weights){
    reg_type regularizer(alpha);
    double *pseudoWeights;
    double *norm, *host_norm;
    host_norm = (double*)malloc(sizeof(double)*num_weights);
    hipError_t err;
    err = hipMalloc((double **)&pseudoWeights, sizeof(double)*num_weights);
    printf(err == hipSuccess ? "." : "Memory allocation failed\n");
    fillMatrix<<<num_weights+1, 1>>>(pseudoWeights, init_weights, num_weights, 1);
    err = hipMalloc((double **)&norm, sizeof(double)*num_weights);
    printf(err == hipSuccess ? "." : "Memory allocation failed\n");
    norm = regularizer.calc_gradient(pseudoWeights, num_weights);
    err = hipMemcpy(host_norm, norm, sizeof(double)*num_weights, hipMemcpyDeviceToHost);
    printf(err == hipSuccess ? "." : "Memory copy failed\n");
    for(int i = 0; i < num_weights; i++){
        printf("\n%f", host_norm[i]);
    }
    printf("\n\n");
    free(host_norm);
    hipFree(pseudoWeights);
    hipFree(norm);
}
#endif
void DenseTest(){
    hipError_t err;
    double *mat, *output;
    double *layer_output;
    double *backward_output;
    int in = 8, out = 5;
    double *temp_input = (double*) malloc(in*sizeof(double));
    
    err = hipMalloc((void**)&mat, in*sizeof(double));
    if(err != hipSuccess)printf("Error allocating memory for mat\n");
    output = (double*)malloc(out*sizeof(double));
    fillMatrix<<<128, 1>>>(mat, 1, in, 1);
    Dense layer(in, out);
    layer_output = layer.forward(mat);
    backward_output = layer.backward(layer_output);
    err = hipMemcpy(output, layer_output, out*sizeof(double), hipMemcpyDeviceToHost);
    if(err != hipSuccess)printf("Error copying output\n");
    err = hipMemcpy(temp_input, backward_output, in*sizeof(double), hipMemcpyDeviceToHost);
    if(err != hipSuccess)printf("Error copying input\n");
    for(int i = 0; i < in; i++){
        printf("%f ", temp_input[i]);
    }
    printf("\n");
    for(int i = 0; i < out; i++){
        printf("%f\n", output[i]);
    }
    free(output);
    hipFree(mat);
}