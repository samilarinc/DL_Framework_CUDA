#include "hip/hip_runtime.h"
#include "headers/L1.cuh"
#include<stdio.h>

#define dabs(x) ((x) < 0.0 ? -(x) : (x))
#define sign(x) ((x) > 0 ? 1 : -1)
#ifndef div2ceil
#define div2ceil(x) (((x) + 1) / 2)
#endif

__global__ void L1_norm(double* w, int size, double* temp){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size){
        temp[idx] = dabs(w[idx]);
    }
}

__global__ void grad(double* w, int size, double* temp, double* alpha){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size){
        temp[idx] = sign(w[idx]) * alpha[0];
    }
}

__global__ void sum_temp(double* temp, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < ((int)size/2)){
        temp[idx] = temp[idx] + temp[size-idx-1];
    }
}

__global__ void mul_alpha(double* temp, double* alpha){
    temp[0] = temp[0] * alpha[0];
}

L1::L1(double alpha, int max_size){
    hipError_t err = hipMalloc((void**)&temp, sizeof(double) * max_size);
    if(err != hipSuccess)printf("Error allocating temp memory\n");
    err = hipMalloc((double **)&dev_alpha, sizeof(double));
    if(err != hipSuccess)printf("Error allocating alpha memory\n");
    err = hipMemcpy(dev_alpha, &alpha, sizeof(double), hipMemcpyHostToDevice);
    if(err != hipSuccess)printf("Error copying alpha\n");
    err = hipMemset(temp, 0, sizeof(double) * max_size);
    if(err != hipSuccess)printf("Error setting temp memory\n");
}

L1::~L1(){
    hipFree(temp);
    hipFree(dev_alpha);
}

double* L1::norm(double* weights, int size){
    L1_norm<<<size+1, 1>>>(weights, size, this->temp);
    while(size > 1){
        sum_temp<<<size+1, 1>>>(this->temp, size);
        size = div2ceil(size);
    }
    mul_alpha<<<1, 1>>>(this->temp, dev_alpha);
    return this->temp;                            /// Only the first element is used
}

double* L1::calc_gradient(double* weights, int size){
    grad<<<size+1, 1>>>(weights, size, this->temp, dev_alpha);
    return this->temp;
}