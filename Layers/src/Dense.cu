#include "hip/hip_runtime.h"
#include "headers/Dense.cuh"
#include <stdio.h>

// #define __debug_backward

__global__ void initialize(double *A, int N, double constant) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        A[i] = constant;
    }
}

__global__ void dense_forward(double* input, double* weight, double* bias, double* out, int in_h, int in_w, int w_w){ // N is the number of columns of A
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(row < in_h && col < w_w){
        double sum = 0;
        for(int k = 0; k < in_w; k++) {
            sum += input[row*in_w+k] * weight[k*w_w+col];
        }
        out[row*w_w+col] = sum + bias[col];
    }
}

__global__ void dot_T_first(double *error, double *weight, double *out, int in_h, int in_w, int w_h, int w_w){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(row < in_w && col < w_w){
        double sum = 0;
        for(int k = 0; k < in_h; k++) {
            sum += error[row*in_h + k] * weight[col*w_h + k];
        }
        out[row*w_w+col] = sum;
    }
}

__global__ void dot_T_sec(double *error, double *weight, double *out, int in_h, int in_w, int w_h, int w_w){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(row < in_h && col < w_h){
        double sum = 0;
        for(int k = 0; k < in_w; k++) {
            sum += error[row*in_w+k] * weight[col*w_w+k];
        }
        out[row*w_h+col] = sum;
    }
}

__global__ void sum_bias(double *error, double *db, int in_h, int in_w){ // CAN BE OPTIMIZED!!!
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < in_h){
        double sum = 0;
        for(int i = 0; i < in_w; i++){
            sum += error[idx*in_w+i];
        }
        db[idx] = sum;
    }
}

Dense::Dense(int in_size, int out_size, Optimizer *optimizer) : BaseLayer(true){
    this->in_size = in_size;
    this->out_size = out_size;
    this->weight_size = in_size * out_size;
    this->w_optimizer = optimizer;
    if (optimizer != NULL) {
        Optimizer *temp_opt = optimizer->clone();
        Regularizer *temp_reg = optimizer->regularizer->clone();
        temp_opt->set_regularizer(temp_reg);
        this->b_optimizer = temp_opt;
    }
    else {
        this->b_optimizer = NULL;
    }
    hipError_t err;
    err = hipMalloc((double **)&this->weights, in_size * out_size * sizeof(double));
    if (err != hipSuccess)printf("Error allocating memory for weights\n");
    initialize<<<128, 1>>>(this->weights, in_size * out_size, 1);
    err = hipMalloc((double **)&this->bias, out_size * sizeof(double));
    if (err != hipSuccess)printf("Error allocating memory for biases\n");
    initialize<<<128, 1>>>(this->bias, out_size, 0);
    err = hipMalloc((double **)&this->output, out_size * sizeof(double));
    if (err != hipSuccess)printf("Error allocating memory for output\n");
    err = hipMemset(this->output, 0, out_size * sizeof(double));
    if (err != hipSuccess)printf("Error setting output to 0\n");
    err = hipMalloc((double **)&last_input, in_size * sizeof(double));
    if(err != hipSuccess)printf("Error allocating memory for last input\n");
    err = hipMalloc((double **)&dW, in_size * out_size * sizeof(double));
    if(err != hipSuccess)printf("Error allocating memory for dw\n");
    err = hipMalloc((double **)&dx, in_size * sizeof(double));
    if(err != hipSuccess)printf("Error allocating memory for dx\n");
    err = hipMalloc((double **)&db, out_size * sizeof(double));
    if(err != hipSuccess)printf("Error allocating memory for db\n");
}

Dense::~Dense() {
    hipFree(this->weights);
    hipFree(this->bias);
    hipFree(this->output);
    hipFree(this->last_input);
    hipFree(this->dW);
    hipFree(this->dx);
    hipFree(this->db);
}

double* Dense::forward(double *input){
    dim3 dimGrid(32, 32); //change 1 to batch size
    dim3 dimBlock(32, 32);
    hipError_t err = hipMemcpy(this->last_input, input, in_size * sizeof(double), hipMemcpyDeviceToDevice);
    if(err != hipSuccess)printf("Error copying input to last input\n");
    
    #ifdef __debug_forward
    double *temp = (double *)malloc(in_size * out_size * sizeof(double));
    err = hipMemcpy(temp, this->weights, in_size * out_size * sizeof(double), hipMemcpyDeviceToHost);
    if(err != hipSuccess)printf("Error copying weights to host\n");
    printf("Inside dense weights:\n");
    for(int i = 0; i < in_size; i++){
        for(int j = 0; j < out_size; j++){
            printf("%f ", temp[i*out_size+j]);
        }
        printf("\n");
    }
    free(temp);
    

    double *temp3 = (double *)malloc(in_size * sizeof(double));
    err = hipMemcpy(temp, input, in_size * sizeof(double), hipMemcpyDeviceToHost);
    if(err != hipSuccess)printf("Error copying weights to host\n");
    printf("Inside dense input:\n");
    for(int i = 0; i < in_size; i++){
            printf("%f ", temp[i]);
        }
    printf("\n");
    free(temp3);
    #endif
    dense_forward<<<dimBlock, dimGrid>>>(input, this->weights, this->bias, this->output, 1, this->in_size, this->out_size); // 1 is the number of rows of A, might depend on the batch size
    #ifdef __debug
    double *temp2 = (double *)malloc(out_size * sizeof(double));
    err = hipMemcpy(temp2, this->output, out_size * sizeof(double), hipMemcpyDeviceToHost);
    if(err != hipSuccess)printf("Error copying weights to host\n");
    printf("Inside dense output:\n");
    for(int i = 0; i < out_size; i++){
            printf("%f ", temp2[i]);
        }
    printf("\n");
    free(temp2);
    #endif // __debug_forward
    return this->output;
}

double* Dense::backward(double *error_tensor){
    dim3 dimGrid(32, 32); 
    dim3 dimBlock(32, 32);
    dot_T_sec<<<dimGrid, dimBlock>>>(error_tensor, this->weights, this->dx, 1, this->out_size, this->in_size, this->out_size);
    dot_T_first<<<dimGrid, dimBlock>>>(this->last_input, error_tensor, this->dW, 1, this->in_size, 1, this->out_size);
    sum_bias<<<dimGrid, dimBlock>>>(error_tensor, this->db, 1, this->out_size);
    if(w_optimizer != NULL){
        w_optimizer->step(this->weights, this->dW, this->weight_size);
        b_optimizer->step(this->bias, this->db, this->out_size);
    }
    #ifdef __debug_backward
    double *temp = (double *)malloc(in_size * out_size * sizeof(double));
    hipError_t err = hipMemcpy(temp, this->dW, in_size * out_size * sizeof(double), hipMemcpyDeviceToHost);
    if(err != hipSuccess)printf("Error copying weights to host\n");
    printf("Inside dense dW:\n");
    for(int i = 0; i < in_size; i++){
        for(int j = 0; j < out_size; j++){
            printf("%f ", temp[i*out_size+j]);
        }
        printf("\n");
    }
    free(temp);
    #endif // __debug_backward
    return this->dx;
}